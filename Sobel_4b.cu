#include "hip/hip_runtime.h"
/**************************************************************************
*   文件名：Sobel_4b.cu
*   作者：  孙霖(SC19023100)(Seafood)
*   说明：  cudau作业之Sobel算子边缘检测第四版
*   将数组存入纹理内存
*   
****************************************************************************/
//----------------------------头文件包含和空间声明------------------------------------
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <opencv2/opencv.hpp>
#include <iostream>
#include <time.h>

typedef unsigned char uchar;

//命名空间
using namespace std;
using namespace cv;

//#define DIM 2048
//----------------------------全局变量---------------------------------------------
Mat g_sobelGradient_X, g_sobelGradient_Y;   //opencv包使用的全局变量
Mat g_sobelAbsGradient_X, g_sobelAbsGradient_Y; //opencv包使用的全局变量
int g_sobelKernelSize = 1;


Mat dst_Cpu, g_gaussImage, g_grayImage, g_dstImage; //图像读取+高斯滤波

int g_imgHeight, g_imgWidth;    //图像的大小Size


int sobel_x[3][3];
int sobel_y[3][3];
int number = 1025;
//cuda的常量内存
__constant__ int dev_sobel_x[3][3];
__constant__ int dev_sobel_y[3][3];

//纹理内存
texture<unsigned char, 1> texIn;
/**
*@author：Seafood
*@name：sobelInCuda()
*@return:void
*@function：使用Cuda对图像进行Sobel边缘检测
*@para：*dataIn:输入图像 *dataOut:输出图像 imgHeight：图像的高 imgWidth:图像的宽
*其他要注意的地方
**/
//Sobel算子边缘检测核函数
__global__ void sobelInCuda(unsigned char *dataOut, int imgHeight, int imgWidth)
{
    //用单thread操作
    //int index = threadIdx.x + blockIdx.x * blockDim.x;
    int xIndex = threadIdx.x + blockIdx.x * blockDim.x;
    int yIndex = threadIdx.y + blockIdx.y * blockDim.y;
    int offset = xIndex + yIndex * imgWidth;
    //printf("blockDim: %d, gridDim: %d\n", blockDim.x, gridDim.x);
    //printf("xIndex : %d,yIndex : %d,Index : %d\n",xIndex, yIndex, offset);
    /*
    if(threadIdx.x == 1 && blockIdx.x == 1)
    {
        for(int i = 0;i < 3; i++)
        {
            printf("%d %d %d\n", dev_sobel_y[i][0],dev_sobel_y[i][1],dev_sobel_y[i][2]);
        }
        printf("gpu用的高斯图像%s\n", tex1Dfetch(texIn, 1025*imgWidth + 1025));
    }*/
    
    int tex_00,tex_01,tex_02,tex_10,tex_11,tex_12,tex_20,tex_21,tex_22;
    int Gx = 0;
    int Gy = 0;

    //printf("NonTex： %d \nTex: %d \n", dataIn[offset+ imgWidth* 2], tex1D(texIn, offset+ imgWidth* 2));

    while(offset < (imgHeight - 2) * (imgWidth - 2))
    {
        //纹理内存上下左右
        int down_00 = offset;                   int down_01 = down_00 + 1;  int down_02 = down_00 + 2;
        int down_10 = offset + imgWidth;        int down_11 = down_10 + 1;  int down_12 = down_10 + 2;
        int down_20 = offset + imgWidth * 2;    int down_21 = down_20 + 1;  int down_22 = down_20 + 2;

        tex_00 = tex1Dfetch(texIn, down_00);    tex_01 = tex1Dfetch(texIn, down_01);    tex_02 = tex1Dfetch(texIn, down_02);
        tex_10 = tex1Dfetch(texIn, down_10);    tex_11 = tex1Dfetch(texIn, down_11);    tex_12 = tex1Dfetch(texIn, down_12);
        tex_20 = tex1Dfetch(texIn, down_20);    tex_21 = tex1Dfetch(texIn, down_21);    tex_22 = tex1Dfetch(texIn, down_22);
        
        Gx = dev_sobel_x[0][0] * tex_00 + dev_sobel_x[0][1] * tex_01 + dev_sobel_x[0][2] * tex_02
            + dev_sobel_x[1][0] * tex_10 + dev_sobel_x[1][1] * tex_11 + dev_sobel_x[1][2] * tex_12
            + dev_sobel_x[2][0] * tex_20 + dev_sobel_x[2][1] * tex_21 + dev_sobel_x[2][2] * tex_22;
        Gy = dev_sobel_y[0][0] * tex_00 + dev_sobel_y[0][1] * tex_01 + dev_sobel_y[0][2] * tex_02
            + dev_sobel_y[1][0] * tex_10 + dev_sobel_y[1][1] * tex_11 + dev_sobel_y[1][2] * tex_12
            + dev_sobel_y[2][0] * tex_20 + dev_sobel_y[2][1] * tex_21 + dev_sobel_y[2][2] * tex_22;

        int sum = abs(Gx) + abs(Gy);
        //int sum = Gx;
        if (sum > 255)
        {
            sum = 255; //for best performance
        }
        if (sum < 0) sum = 0;
        dataOut[offset] = (int)sum;
        xIndex += blockDim.x * gridDim.x;
        if( xIndex > imgWidth)
        {
            yIndex += blockDim.y * gridDim.y;
            xIndex = threadIdx.x + blockIdx.x * blockDim.x;
        }
        
        //offset = xIndex + yIndex * blockDim.x * gridDim.x;
        offset = xIndex + yIndex * imgWidth;
        //index = xIndex + yIndex * imgWidth;
    }
}

/**
*@author：Seafood
*@name：sobelCpuPixel()
*@return:void016 x 401
*@function：使用CPU对图像像素进行Sobel边缘检测
*@para：None
*其他要注意的地方
**/
//Sobel算子边缘检测CPU函数
void sobelCpuPixel()
{
    clock_t begin_time, end_clock;
    begin_time = clock();

    //CPU用
    Mat img = g_gaussImage;
    Mat newimg = img;
    for (int j = 0; j<img.rows-2; j++)
    {
     for (int i = 0; i<img.cols-2; i++)
     {
         int pixval_x =
         (sobel_x[0][0] * (int)img.at<uchar>(j,i)) + (sobel_x[0][1] * (int)img.at<uchar>(j+1,i)) + (sobel_x[0][2] * (int)img.at<uchar>(j+2,i)) +
         (sobel_x[1][0] * (int)img.at<uchar>(j,i+1)) + (sobel_x[1][1] * (int)img.at<uchar>(j+1,i+1)) + (sobel_x[1][2] * (int)img.at<uchar>(j+2,i+1)) +
         (sobel_x[2][0] * (int)img.at<uchar>(j,i+2)) + (sobel_x[2][1] * (int)img.at<uchar>(j+1,i+2)) + (sobel_x[2][2] * (int)img.at<uchar>(j+2,i+2));
         
         int pixval_y =
         (sobel_y[0][0] * (int)img.at<uchar>(j,i)) + (sobel_y[0][1] * (int)img.at<uchar>(j+1,i)) + (sobel_y[0][2] * (int)img.at<uchar>(j+2,i)) +
         (sobel_y[1][0] * (int)img.at<uchar>(j,i+1)) + (sobel_y[1][1] * (int)img.at<uchar>(j+1,i+1)) + (sobel_y[1][2] * (int)img.at<uchar>(j+2,i+1)) +
         (sobel_y[2][0] * (int)img.at<uchar>(j,i+2)) + (sobel_y[2][1] * (int)img.at<uchar>(j+1,i+2)) + (sobel_y[2][2] * (int)img.at<uchar>(j+2,i+2));
         
         int sum = abs(pixval_x) + abs(pixval_y);
         //int sum = pixval_x;
         if (sum > 255)
         {
             sum = 255; //for best performance
         }
         if (sum < 0) sum = 0;
         newimg.at<uchar>(j,i) = sum;
     }
    }
    end_clock = clock();
    cout << "CPU对像素操作运行时间为: " << static_cast<double>(end_clock - begin_time) / CLOCKS_PER_SEC*1000 << "ms" << endl;//输出运行时间为毫秒
    cvNamedWindow("processed by CPU in Paxel", 0);
    resizeWindow("processed by CPU in Paxel", 800, 600);
    imshow("processed by CPU in Paxel", newimg);
}

/**
*@author：Seafood
*@name：sobelCpuPackage
*@return:void
*@function：使用CPU的OPENCV包对图像进行Sobel边缘检测
*@para：None
*其他要注意的地方
**/
//Sobel算子边缘检测CPU函数

void sobelCpuPackage(int, void*)
{
    clock_t begin_time, end_clock;
    begin_time = clock();
    //求x方向梯度
    Sobel(g_grayImage, g_sobelGradient_X, CV_16S, 1, 0, (2*g_sobelKernelSize + 1), 1, 1, BORDER_DEFAULT);
    convertScaleAbs( g_sobelGradient_X, g_sobelAbsGradient_X);

    //求Y方向梯度
    Sobel(g_grayImage, g_sobelGradient_Y, CV_16S, 1, 0, (2*g_sobelKernelSize + 1), 1, 1, BORDER_DEFAULT);
    convertScaleAbs( g_sobelGradient_Y, g_sobelAbsGradient_Y);

    //合并梯度
    addWeighted(g_sobelAbsGradient_X, 0.5, g_sobelAbsGradient_Y, 0.5,0,dst_Cpu );
    end_clock = clock();
    cout << "CPU对使用OPENCV包操作运行时间为: " << static_cast<double>(end_clock - begin_time) / CLOCKS_PER_SEC*1000 << "ms" << endl;//输出运行时间为毫秒
    cvNamedWindow("CPU处理效果图(OPENCV包)", 0);
    resizeWindow("CPU处理效果图(OPENCV包)", 800, 600);
    imshow("CPU处理效果图(OPENCV包)", dst_Cpu);
}

/**
*@author：Seafood
*@name：pictureInit(void)
*@return:void
*@function：图像读取、高斯滤波和初始化
*@para：None
*其他要注意的地方
**/

//图像读入
void pictureInit(void)
{
    //读入图像
    g_grayImage = imread("rmpicture.png", 0);

    //显示原图
    cvNamedWindow("originimage", 0);
    resizeWindow("originimage", 800, 600);
    imshow("originimage", g_grayImage);
    g_imgHeight = g_grayImage.rows;
    g_imgWidth = g_grayImage.cols;

    // asigning values to sobel x direction
    sobel_x[0][0] = -1; sobel_x[0][1] = 0; sobel_x[0][2] =1;
    sobel_x[1][0] = -2; sobel_x[1][1] = 0; sobel_x[1][2] =2;
    sobel_x[2][0] = -1; sobel_x[2][1] = 0; sobel_x[2][2] =1;
    // asigning values to sobel y direction
    sobel_y[0][0] = -1; sobel_y[0][1] = -2; sobel_y[0][2] = -1;
    sobel_y[1][0] = 0; sobel_y[1][1] = 0; sobel_y[1][2] = 0;
    sobel_y[2][0] = 1; sobel_y[2][1] = 2; sobel_y[2][2] = 1;
    printf(" picture size is %d x %d \n", g_imgHeight, g_imgWidth);
    //高斯滤波
    GaussianBlur(g_grayImage, g_gaussImage, Size(3,3), 0, 0, BORDER_DEFAULT);
    cvNamedWindow("gaussimage", 0);
    resizeWindow("gaussimage", 800, 600);
    imshow("gaussimage", g_gaussImage);
}
/**
*@author：Seafood
*@name：sobelGPUPixel()
*@return:void
*@function：调用GPU对图像处理并计算时间
*@para：None
*其他要注意的地方
**/
//GPU处理函数
void sobelGPUPixel()
{
    //定义变量
    unsigned char *dev_in;
    unsigned char *dev_out;

    Mat dst_Gpu(g_imgHeight, g_imgWidth, CV_8UC1, Scalar(0));
    //创建时间用于计算
    hipEvent_t start, stop;
    hipEventCreate( &start );
    hipEventCreate( &stop );
    //开始时间点记录
    hipEventRecord( start, 0 );
    


    //定义数组
    //申请内存
    hipMalloc((void**)&dev_in, g_imgHeight * g_imgWidth * sizeof(unsigned char));
    hipMalloc((void**)&dev_out, g_imgHeight * g_imgWidth * sizeof(unsigned char));
    //纹理内存绑定
    hipBindTexture(NULL, texIn, dev_in, g_imgHeight * g_imgWidth * sizeof(unsigned char));
    //导入内存
    hipMemcpy(dev_in, g_gaussImage.data, g_imgHeight * g_imgWidth * sizeof(unsigned char), hipMemcpyHostToDevice);
    
    //数组导入
    hipMemcpyToSymbol(HIP_SYMBOL(dev_sobel_x), sobel_x, sizeof(sobel_x) );
    hipMemcpyToSymbol(HIP_SYMBOL(dev_sobel_y), sobel_y, sizeof(sobel_y) );
    
    //调用核函数
    //定义block thread范围
    dim3 blocks((int)((g_imgWidth+31)/32), (int)(g_imgHeight+31)/32);
    //dim3 blocks(4, 4);
    dim3 threads(16, 16);
    //单block 单thread
    sobelInCuda<<<4,4>> >(dev_out, g_imgHeight, g_imgWidth);
    
    //导出处理
    hipMemcpy(dst_Gpu.data, dev_out, g_imgHeight * g_imgWidth * sizeof(unsigned char), hipMemcpyDeviceToHost);

    //停止时间点
    hipEventRecord( stop, 0 );
    hipEventSynchronize( stop );

    //释放内存
    hipFree(dev_in);
    hipFree(dev_out);
    hipFree(dev_sobel_x);
    hipFree(dev_sobel_y);


    //计算GPU所用时间
    float elapsedTime;
    hipEventElapsedTime( &elapsedTime, start, stop );
    printf( "GPU对像素操作运行时间为: %.1f ms \n", elapsedTime );
    hipEventDestroy( start );
    hipEventDestroy( stop );

    //显示处理后的图像
    cvNamedWindow("GPU处理后的图像", 0);
    resizeWindow("GPU处理后的图像", 800, 600);
    imshow("GPU处理后的图像", dst_Gpu);
}
/**
*@author：Seafood
*@name：main(int argc, char *argv[], char **env)
*@return:int
*@function：程序入口，主函数
*@para：None
*其他要注意的地方
**/
//main函数
int main(int argc, char *argv[], char **env)
{
    
    //图像读入和处理
    pictureInit();
    //Sobel算子Cpu package实现
    
    sobelCpuPackage(0,0);
    
    //测试算子
    //Sobel算子Cpu 对像素操作实现
    sobelCpuPixel();

    //Sobel算子GPU操作实现
    sobelGPUPixel();
    //结束展示
    while((char)waitKey(0) != 'q' )
    {

    }
    
    return 0;
}